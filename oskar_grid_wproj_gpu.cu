#include "hip/hip_runtime.h"
// (C) 2012  John Romein/ASTRON

// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.

// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.

// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.

#if defined __AVX__
#include <immintrin.h>
#endif

#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <sstream>

#include "Common.h"
#include "Defines.h"

#if ORDER == ORDER_W_OV_OU_V_U
typedef REAL2 SupportType[W_PLANES][OVERSAMPLE_V][OVERSAMPLE_U][SUPPORT_V][SUPPORT_U];
#elif ORDER == ORDER_W_V_OV_U_OU
typedef REAL2 SupportType[W_PLANES][SUPPORT_V][OVERSAMPLE_V][SUPPORT_U][OVERSAMPLE_U];
#endif

typedef REAL2 GridType[GRID_V][GRID_U][POLARIZATIONS];

#if UVW_REORDERED
typedef REAL3 UVWtype[BASELINES][TIMESTEPS][CHANNELS];
#else
typedef REAL3 UVWtype[TIMESTEPS][BASELINES][CHANNELS];
#endif

#if VIS_REORDERED
typedef REAL2 VisibilitiesType[BASELINES][TIMESTEPS][CHANNELS][POLARIZATIONS];
#else
typedef REAL2 VisibilitiesType[TIMESTEPS][BASELINES][CHANNELS][POLARIZATIONS];
#endif

unsigned nrThreads;

__device__ void atomicAdd(REAL2 *ptr, REAL2 sumXX)
{
    atomicAdd(&ptr[0].x, sumXX.x);
    atomicAdd(&ptr[0].y, sumXX.y);
}

__device__ void addSupportPixel(REAL2 &sum, REAL2 supportPixel, REAL2 vis)
{

    # if FAKE_VIS_VALUES==1
    vis.x=1.0;
    vis.y=0.0;
    # endif

    sum.x += supportPixel.x * vis.x;
    sum.y += supportPixel.x * vis.y;
    sum.x -= supportPixel.y * vis.y;
    sum.y += supportPixel.y * vis.x;
}

__shared__ int4   shared_info[TIMESTEPS][CHANNELS];
__shared__ REAL2 shared_vis[TIMESTEPS][CHANNELS][POLARIZATIONS];

__device__ void loadIntoSharedMem(const VisibilitiesType visibilities,
				  const UVWtype uvw,
				  const uint2 supportPixelsUsed[BASELINES],
                  unsigned full_grid_size)
{
    unsigned bl = blockIdx.x;

    uint2 supportSize = supportPixelsUsed[bl];

    #if UVW_REORDERED
        int zCoord = roundf(uvw[bl][0][0].z);
    #else
        int zCoord = roundf(uvw[0][bl][0].z);
    #endif
    
    int grid_centre_u = full_grid_size/2 - TRIMMED_REGION_OFFSET_U + 1;
    int grid_centre_v = full_grid_size/2 - TRIMMED_REGION_OFFSET_V + 1;

    //for (unsigned ch = threadIdx.x; ch < CHANNELS * TIMESTEPS; ch += blockDim.x)
    for (int ch = threadIdx.x; ch < CHANNELS * TIMESTEPS; ch += blockDim.x)
    {
        #if UVW_REORDERED
            REAL3   coords = uvw[bl][0][ch]; // coords = {u,v,w}
        #else
            REAL3   coords = uvw[ch][bl][0]; // coords = {u,v,w}
        #endif

        // the kernel grid is shifted by half a kernel grid cell in relation to the global grid

        int u_int  = __float2int_rd(coords.x);
        int v_int  = __float2int_rd(coords.y);

        //printf("coords: %f %f, u_int %d v_int %d\n", coords.x, coords.y, u_int+TRIMMED_REGION_OFFSET_U+supportSize.x/2, v_int+TRIMMED_REGION_OFFSET_V+supportSize.y/2);

        //if (u_int>2160 && u_int<2166 && v_int<3202 && v_int>3195) printf("blockId: %d\n", blockIdx.x);
        //if (u_int==2164 && v_int==3198) printf("blockId: %d\n", blockIdx.x);

        coords.x += (1.0/OVERSAMPLE_U)/2.0;
        coords.y += (1.0/OVERSAMPLE_V)/2.0;
        int kernel_u_int  = __float2int_rd(coords.x);
        int kernel_v_int  = __float2int_rd(coords.y);

        REAL u_frac, v_frac;
        u_frac = (coords.x - kernel_u_int);
        v_frac = (coords.y - kernel_v_int);

        unsigned u_off, v_off;
        u_off = (unsigned)OVERSAMPLE_U*u_frac;
        v_off = (unsigned)OVERSAMPLE_V*v_frac;

        // convert oversample index to offset
	unsigned additional_kernel_offset = 0;
        u_off = (OVERSAMPLE_U-u_off)%4;
        v_off = (OVERSAMPLE_V-v_off)%4;
	if (u_off==0 && kernel_u_int==u_int){
		additional_kernel_offset = 1;
	}
	if (v_off==0 && kernel_v_int==v_int){
		additional_kernel_offset += SUPPORT_U;
	}

    u_int += grid_centre_u;
    v_int += grid_centre_v;
        
#if ORDER == ORDER_W_OV_OU_V_U
        unsigned uv_frac_w_offset = (unsigned) zCoord * SUPPORT_V * SUPPORT_U * OVERSAMPLE_V * OVERSAMPLE_U + SUPPORT_U * SUPPORT_V * (OVERSAMPLE_U * v_off + (unsigned) u_off) + additional_kernel_offset; // starting index of correct kernel
#elif ORDER == ORDER_W_V_OV_U_OU
        unsigned uv_frac_w_offset = (unsigned) zCoord * SUPPORT_V * OVERSAMPLE_V * SUPPORT_U * OVERSAMPLE_U + (unsigned) (OVERSAMPLE_V * v_frac) * SUPPORT_U * OVERSAMPLE_U + (unsigned) (OVERSAMPLE_U * u_frac);
#endif
        shared_info[0][ch] = make_int4(-u_int % supportSize.x, -v_int % supportSize.y, uv_frac_w_offset, u_int + GRID_U * v_int);
    }

    //for (unsigned i = threadIdx.x; i < CHANNELS * TIMESTEPS * POLARIZATIONS; i += blockDim.x)
    for (int i = threadIdx.x; i < CHANNELS * TIMESTEPS * POLARIZATIONS; i += blockDim.x){
#if VIS_REORDERED
        ((REAL2 *) shared_vis)[i] = ((REAL2 *) visibilities[bl])[i];
#else
        ((REAL2 *) shared_vis)[i] = ((REAL2 *) visibilities[i])[bl];
#endif
    }
}


__device__ void convolve(GridType grid,
			const SupportType support,
			const uint2 supportPixelsUsed[BASELINES], double *norm)
{
    unsigned bl	= blockIdx.x;
    uint2 supportSize = supportPixelsUsed[bl];
    #if CALCULATE_NORM
        double norm_local=0;
    #endif
//	if (bl!=400) return;

    for (int i = supportSize.x * supportSize.y - threadIdx.x - 1; i >= 0; i -= blockDim.x)
    {
        int box_u = - (i % supportSize.x);
        int box_v = - (i / supportSize.x);
        REAL2 sumXX = MAKE_REAL2(0, 0);
        unsigned grid_point = threadIdx.x;// does this cause (0,0) to be added when t=0?
        

    	//for (unsigned ch = 0; ch < CHANNELS * TIMESTEPS; ch++)
    	for (unsigned ch = 0; ch < 1 * TIMESTEPS; ch++)
        {
            // info = { x=-u offset of box from subgrids, y=-v offset of box from subgrids, 
            // z=start index of correct wkernel, w=index of vis-(2*wsupport+1)/2 in global grid 
            // (ie index of beginning of kernel for that visibility)}
        	int4 info = shared_info[0][ch]; 
        	int my_support_u = box_u + info.x;
        	int my_support_v = box_v + info.y;

        	if (my_support_u < 0)
        	    my_support_u += supportSize.x;

        	if (my_support_v < 0)
        	    my_support_v += supportSize.y;

            // thread's index in wkernel
        	unsigned index_u = my_support_u;
        	unsigned index_v = my_support_v;

#if ORDER == ORDER_W_OV_OU_V_U
    	    unsigned supportIndex = index_u + SUPPORT_U * index_v + info.z;
#elif ORDER == ORDER_W_V_OV_U_OU
    	    unsigned supportIndex = OVERSAMPLE_U * index_u + OVERSAMPLE_V * SUPPORT_U * OVERSAMPLE_U * index_v + info.z;
#endif
    	    REAL2 supportPixel;
            # if FAKE_KERNEL_VALUES==1
    	    supportPixel = MAKE_REAL2(1.0, 0.0);
            # else
            supportPixel= support[0][0][0][0][supportIndex];
            # endif

        	unsigned new_grid_point = my_support_u + GRID_U * my_support_v + info.w;

            if (new_grid_point != grid_point)
            {
                atomicAdd(&grid[0][grid_point][0], sumXX);
                sumXX = MAKE_REAL2(0, 0);
                grid_point = new_grid_point;
            }
            addSupportPixel(sumXX, supportPixel, shared_vis[0][ch][0]);
            #if CALCULATE_NORM
                norm_local += supportPixel.x;
            #endif
        }
        atomicAdd(&grid[0][grid_point][0], sumXX);
    }
   
    #if CALCULATE_NORM 
        atomicAdd(norm, norm_local);
    #endif
}


//#if MODE == MODE_SIMPLE || MODE == MODE_OVERSAMPLE
#define MIN(A,B)			((A) < (B) ? (A) : (B))
#define NR_THREADS_PER_BLOCK		MIN(SUPPORT_U * SUPPORT_V, 1024)
#define MIN_BLOCKS_PER_MULTIPROCESSOR	(2048 / NR_THREADS_PER_BLOCK)
__global__ __launch_bounds__(NR_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MULTIPROCESSOR)
//#else
//__global__ __launch_bounds__(1024, 1)
//#endif
void addToGrid(GridType grid,
	       const SupportType support,
	       const VisibilitiesType visibilities,
	       const UVWtype uvw,
	       const uint2 supportPixelsUsed[BASELINES],
           unsigned full_grid_size,
           double *norm)
{
    //if (blockIdx.x!=120776) return;
    //if (blockIdx.x!=108953 && blockIdx.x!=109367 && blockIdx.x!=109368 && blockIdx.x!=109574 && blockIdx.x!=109571) return;
    //if (blockIdx.x!=109571) return;
    loadIntoSharedMem(visibilities, uvw, supportPixelsUsed, full_grid_size);
    __syncthreads();
    convolve(grid, support, supportPixelsUsed, norm);
}



void initUVW(UVWtype uvw, uint2 supportPixelsUsed[BASELINES], const REAL frequencies[CHANNELS], unsigned block,
	unsigned full_grid_size, const double cell_size_rad, const double w_scale, 
    const REAL *uu, const REAL *vv, const REAL *ww, const int *supportSize)
{
    // we only process the trimmed region of the global grid which contains visibilities
    // input uvw coordinates will have their origin at the global grid centre
    //int grid_centre_u = full_grid_size/2 - TRIMMED_REGION_OFFSET_U + 1;
    //int grid_centre_v = full_grid_size/2 - TRIMMED_REGION_OFFSET_V + 1;
    printf("block: %d\n", block);

    for (unsigned bl = 0; bl < BASELINES; bl ++) {
        for (unsigned time = 0; time < TIMESTEPS; time ++) {

            // uu[TIMESTEPSxBLOCKS][BASELINES]
	        const int currentUVWIndex = (block*TIMESTEPS + time)*BASELINES + bl;

			REAL scaled_u = full_grid_size*cell_size_rad * -uu[currentUVWIndex];
			REAL scaled_v = full_grid_size*cell_size_rad * vv[currentUVWIndex];
            		REAL w = ww[currentUVWIndex];
        
			REAL scaled_w = sqrt(fabs(w * w_scale)); 
            if (time==0){
				// init support
				supportPixelsUsed[bl].x = supportSize[(int)roundf(scaled_w)]*2 + 1; 
				supportPixelsUsed[bl].y = supportSize[(int)roundf(scaled_w)]*2 + 1; 
            }

            // seperate kernels are stored for w<0 and w>0
            if (w>0) scaled_w = scaled_w + W_PLANES/2;
            else scaled_w = W_PLANES/2 - scaled_w; 

            // use only one channel
            #if UVW_REORDERED
                uvw[bl][time][0] = MAKE_REAL3(
                        scaled_u - supportPixelsUsed[bl].x / 2.0f,
                        scaled_v - supportPixelsUsed[bl].y / 2.0f,
                        scaled_w
                );
            #else
                uvw[time][bl][0] = MAKE_REAL3(
                        scaled_u - supportPixelsUsed[bl].x / 2.0f,
                        scaled_v - supportPixelsUsed[bl].y / 2.0f,
                        scaled_w
                );
            #endif
        }
    }
}

REAL2 getSupportVal(const REAL *supportIn, const int conv_size_half, int ou, int ov, int u, int v, int w, int wNegative){
    int supportOffset = w * conv_size_half*conv_size_half;
    int supportIndex = supportOffset + (v*OVERSAMPLE_V+ov)*conv_size_half + u*OVERSAMPLE_U + ou;
    REAL real = supportIn[supportIndex*2];
    REAL imag = supportIn[supportIndex*2+1];
    if (!wNegative) imag *= -1.0;
    return MAKE_REAL2(real, imag);
}

void initSupport(SupportType support, const REAL* supportIn, const int* supportSize, const int conv_size_half)
{
    // SUPPORT_U, SUPPORT_V := 2*maxWSupport+1
    for (int wIndex = 0; wIndex < W_PLANES; wIndex ++){
        int w = wIndex-W_PLANES/2;
        int wNegative = w < 0 ? 1 : 0;
        w = abs(w);
        int wsupport = supportSize[w];
        int padding = SUPPORT_U - (2*wsupport+2);
        int uOut, vOut, u, v;
        for (int ov = 0; ov < OVERSAMPLE_V; ov ++){
            for (int ou = 0; ou < OVERSAMPLE_U; ou ++){
		vOut=0;
                // top two quadrants of kernel
                for (v=wsupport; v>=0; v--){
                    uOut = 0;
                    // left quadrant of kernel
                    for (u=wsupport; u>=0; u--){
                        support[wIndex][ov][ou][vOut][uOut++] = getSupportVal(supportIn, conv_size_half,
                            (OVERSAMPLE_U-ou-OVERSAMPLE_U/2), (OVERSAMPLE_V-ov-OVERSAMPLE_V/2), u, v, w, wNegative);
                    }
                    // right quadrant of kernel
                    uOut = uOut-1;
                    for (u=0; u<=wsupport+1; u++){
                        if (u==0 && ou<OVERSAMPLE_U/2) { uOut++; continue; }
			if (u==wsupport+1 && ou>0) {uOut++; continue; }
                        support[wIndex][ov][ou][vOut][uOut++] = getSupportVal(supportIn, conv_size_half,
                            ou-OVERSAMPLE_U/2, (OVERSAMPLE_V-ov-OVERSAMPLE_V/2), u, v, w, wNegative);
                    }
                    // padding to 2*MAX_W_SUPPORT+1
                    for (u=0; u<padding; u++) {
                        support[wIndex][ov][ou][vOut][uOut++]=MAKE_REAL2(0.,0.);
                    }

                    vOut++;
                }
		vOut = vOut-1;
                // bottom two quadrants of kernel
                for (v=0; v<=wsupport+1; v++){
                    if (v==0 && ov<OVERSAMPLE_V/2) { vOut++; continue; }
		    if (v==wsupport+1 && ov>0) {uOut++; continue; }
                    uOut = 0;
                    // left quadrant of kernel
                    for (u=wsupport; u>=0; u--){
                        support[wIndex][ov][ou][vOut][uOut++] = getSupportVal(supportIn, conv_size_half,
                            OVERSAMPLE_U-ou-OVERSAMPLE_U/2, ov-OVERSAMPLE_V/2, u, v, w, wNegative);
                    }
                    uOut = uOut-1;
                    // right quadrant of kernel
                    for (u=0; u<=wsupport+1; u++){
                        if (u==0 && ou<OVERSAMPLE_U/2) { uOut++; continue; }
			if (u==wsupport+1 && ou>0) {uOut++; continue; }
                        support[wIndex][ov][ou][vOut][uOut++] = getSupportVal(supportIn, conv_size_half,
                            ou-OVERSAMPLE_U/2, ov-OVERSAMPLE_V/2, u, v, w, wNegative);
                    }
                    // padding to 2*MAX_W_SUPPORT+1
                    for (u=0; u<padding; u++) {
                        support[wIndex][ov][ou][vOut][uOut++]=MAKE_REAL2(0.,0.);
                    }
                    vOut++;
                }

                // padding to 2*MAX_W_SUPPORT+1
                for (v=0; v<padding; v++) {
                    uOut = 0;
                    for (u=0; u<SUPPORT_U; u++){
                        support[wIndex][ov][ou][vOut][uOut++]=MAKE_REAL2(0.,0.);
                    }
                    vOut++;
                }
	    }
        }
    }
}

void initFrequencies(REAL frequencies[CHANNELS])
{
    for (unsigned ch = 0; ch < CHANNELS; ch ++)
        frequencies[ch] = 59908828.7353515625 + 12207.03125 * ch;
}


void initVisibilities(VisibilitiesType visibilities, const REAL *inputVis, unsigned block)
{
    // reorder input visibility data
    // visibilities[BASELINES][TIMESTEPS][CHANNELS][POLARIZATIONS]
    // inputVis[TIMESTEPSxBLOCKS][BASELINES][2] (as 1D array)
    REAL real, imag;
    REAL2 vis;
    for (int bl=0; bl<BASELINES; bl++){
        for (int t=0; t<TIMESTEPS; t++){
            // use one channel and one polarisation
            real = inputVis[ (block*TIMESTEPS+t)*BASELINES*2 + bl*2 ];
            imag = inputVis[ (block*TIMESTEPS+t)*BASELINES*2 + bl*2 + 1 ];
            vis = MAKE_REAL2(real, imag);
            #if VIS_REORDERED
                visibilities[bl][t][0][0] = vis;
            #else
                visibilities[t][bl][0][0] = vis;
            #endif
        }
    }
}

void printWorkLoad(uint2 supportPixelsUsed[BASELINES])
{
    unsigned long long gridPointUpdates = 0;

    for (unsigned bl = 0; bl < BASELINES; bl ++)
        gridPointUpdates += TIMESTEPS * CHANNELS * POLARIZATIONS * supportPixelsUsed[bl].x * supportPixelsUsed[bl].y;

#pragma omp critical (cout)
    std::cout << "gridPointUpdates = " << gridPointUpdates << std::endl;
}

void printGrid(const GridType grid, const char *who)
{
//std::cout << * (unsigned long long *) &grid[4095][4095][0] << " out of " << * (unsigned long long *) &grid[4095][4095][1] << " (" << 100.0 * * (unsigned long long *) &grid[4095][4095][0] / * (unsigned long long *) &grid[4095][4095][1] << "%)" << std::endl;
    unsigned count_v = 0;
    double2 sum = make_double2(0, 0);
    for (unsigned v = 0; v < GRID_V; v ++) {
        unsigned count_u = 0;

        for (unsigned u = 0; u < GRID_U; u ++) {
            if (grid[v][u][0].x != 0 || grid[v][u][0].y != 0) {
	            if (count_u ++ == 0)
	                count_v ++;

                if (count_u < 5 && count_v < 5)
#pragma omp critical (cout)
	               std::cout << who << ": (" << u << ", " << v << "): " << grid[v][u][0] << std::endl;
                sum.x += grid[v][u][0].x;
                sum.y += grid[v][u][0].y;
            }
        }
    }
#pragma omp critical (cout)
    std::cout << "sum = " << sum << std::endl;
}



void initSupportOnHostAndDevice(SharedObject<SupportType> &support, const REAL* conv_func, const int *supportSize,
        const int conv_size_half)
{
    initSupport(*support.hostPtr, conv_func, supportSize, conv_size_half);
    support.copyHostToDevice();
}

void copySupportToCube(SupportType supports, REAL* kernels_cube){
    int outIndex;
    REAL real, imag;
    for (int w=0; w<W_PLANES; w++){
        for (int ov = 0; ov < OVERSAMPLE_V; ov ++){
            for (int ou = 0; ou < OVERSAMPLE_U; ou ++){
                for (int v = 0; v < SUPPORT_V; v ++){
                    for (int u = 0; u < SUPPORT_U; u ++){
                        real = supports[w][ov][ou][v][u].x;
                        imag = supports[w][ov][ou][v][u].y;
                        // separate out oversample offset values for each kernel
                        //outIndex = w*OVERSAMPLE_V*OVERSAMPLE_U*SUPPORT_V*SUPPORT_U +
                         //           ov*OVERSAMPLE_U*SUPPORT_V*SUPPORT_U + ou*SUPPORT_V*SUPPORT_U +
                          //          v*SUPPORT_U + u;

                        // put all oversample offset values in same kernel
                        outIndex =  w*OVERSAMPLE_V*OVERSAMPLE_U*SUPPORT_V*SUPPORT_U + 
                                (v*OVERSAMPLE_V+ov)*SUPPORT_U*OVERSAMPLE_U + u*OVERSAMPLE_U+ou;
                        kernels_cube[2*outIndex] = real;
                        kernels_cube[2*outIndex + 1] = imag;
                    }
                }
            }
        }
    }
}


void oskar_grid_wproj_gpu(const int num_w_planes, const int* supportSize,
        const int oversample, const int conv_size_half,
        const REAL* conv_func, const int num_vis,
        const REAL* uu, 	// uu[TIMESTEPSxBLOCKS][BASELINES]
	const REAL* vv,
        const REAL* ww, 
	const REAL* vis, 	// vis[TIMESTEPSxBLOCKS][BASELINES][2] 
        const REAL* weight, const double cell_size_rad,
        const double w_scale, const int grid_size, size_t* num_skipped,
        double* norm, REAL* gridOut)
{
    int device = 0;

    checkCudaCall(hipSetDevice(device));
    checkCudaCall(hipSetDeviceFlags(hipDeviceMapHost));

    SharedObject<GridType> grids[STREAMS];

    for (unsigned stream = 0; stream < STREAMS; stream ++)
        checkCudaCall(hipMemset(grids[stream].devPtr, 0, sizeof(GridType)));

    SharedObject<SupportType> supports[STREAMS];


    // do we need to copies of support? should be read only
    for (unsigned stream = 0; stream < STREAMS; stream ++)
        initSupportOnHostAndDevice(supports[stream], conv_func, supportSize, conv_size_half);

    // for debugging
    //copySupportToCube(*supports[0].hostPtr, kernels_cube);

    REAL frequencies[CHANNELS];
    initFrequencies(frequencies);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(addToGrid), hipFuncCachePreferShared);

    unsigned nrThreads = NR_THREADS_PER_BLOCK;
    double start = getTime();

#if defined MAP_OBJECTS
    MappedObject<uint2 [BASELINES]> supportPixelsUsed[STREAMS];
    MappedObject<UVWtype> uvw[STREAMS];
    MappedObject<VisibilitiesType> visibilities[STREAMS];
#else
    SharedObject<uint2 [BASELINES]> supportPixelsUsed[STREAMS];
    SharedObject<UVWtype> uvw[STREAMS];
    SharedObject<VisibilitiesType> visibilities[STREAMS];
#endif

    double *dev_norm;
    hipMalloc((void**)&dev_norm, sizeof(double));

    Stream streams[STREAMS];

    printf("num w planes: %d\n", num_w_planes);
    
    printf("timesteps %d blocks %d streams %d bl %d channels %d\n", TIMESTEPS, BLOCKS, STREAMS, BASELINES, CHANNELS);

//#pragma omp critical (cout)
    std::cout << "using " << nrThreads << /*'/' << bestNrThreads <<*/ " threads" << std::endl;
    for (unsigned block = 0; block < BLOCKS; block += STREAMS)
    //for (unsigned block = 86; block < BLOCKS; block += STREAMS)
    {
        for (unsigned stream = 0; stream < STREAMS; stream++)
        {
	        initUVW(*uvw[stream].hostPtr, *supportPixelsUsed[stream].hostPtr, frequencies, block + stream, 
			grid_size, cell_size_rad, w_scale, uu, vv, ww, supportSize);
    	    initVisibilities(*visibilities[stream].hostPtr, vis, block + stream);
        }

    checkCudaCall(hipDeviceSynchronize());
        for (unsigned stream = 0; stream < STREAMS; stream++)
        {
        	visibilities[stream].copyHostToDevice(streams[stream]);
        	uvw[stream].copyHostToDevice(streams[stream]);
        	supportPixelsUsed[stream].copyHostToDevice(streams[stream]);
        }

    checkCudaCall(hipDeviceSynchronize());
        for (unsigned stream = 0; stream < STREAMS; stream++)
        {
	        printWorkLoad(*supportPixelsUsed[stream].hostPtr);
        	addToGrid<<<BASELINES, nrThreads, 0, streams[stream]>>>(
                    *grids[stream].devPtr, *supports[stream].devPtr,
                    *visibilities[stream].devPtr,
                    *uvw[stream].devPtr, *supportPixelsUsed[stream].devPtr,
                    grid_size, dev_norm);
    	   checkCudaCall(hipGetLastError());
        }
    checkCudaCall(hipDeviceSynchronize());
    }
    checkCudaCall(hipDeviceSynchronize());

    hipMemcpy(norm, dev_norm, sizeof(double), hipMemcpyDeviceToHost);
    printf("NORM! %f\n", *norm);
    Event startCopy, finishedCopy;
    startCopy.record();
    grids[0].copyDeviceToHost();
    finishedCopy.record();
    finishedCopy.synchronize();
    int gridOutOffset = TRIMMED_REGION_OFFSET_V*grid_size*2 + TRIMMED_REGION_OFFSET_U*2;
    for (int v=0; v<GRID_V; v++){
        for (int u=0; u<GRID_U; u++){
            gridOut[gridOutOffset + v*grid_size*2 + u*2] =      (*grids[0].hostPtr)[v][u][0].x;
            gridOut[gridOutOffset + v*grid_size*2 + u*2 +1] =   (*grids[0].hostPtr)[v][u][0].y;
        }
    }

    double stop = getTime();
    std::cout << "dev->host copy = " << finishedCopy.elapsedTime(startCopy) << std::endl << "total exec time = " << (stop - start) << std::endl;

    printGrid(*grids[0].hostPtr, "GPU - Cuda");
}

